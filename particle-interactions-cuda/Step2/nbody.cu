#include "hip/hip_runtime.h"
/**
 * @file      nbody.cu
 *
 * @author    Ján Maťufka \n
 *            Faculty of Information Technology \n
 *            Brno University of Technology \n
 *            xmatuf00@stud.fit.vutbr.cz
 *
 * @brief     PCG Assignment 1
 *
 * @version   2024
 *
 * @date      04 October   2023, 09:00 (created) \n
 */

#include <>

#include "nbody.cuh"

/* Constants */
constexpr float G                  = 6.67384e-11f;
constexpr float COLLISION_DISTANCE = 0.01f;
constexpr float FLOAT_MIN          = 1.1754944e-38f;

/**
 * CUDA kernel to calculate new particles velocity and position
 * @param pIn  - particles in
 * @param pOut - particles out
 * @param N    - Number of particles
 * @param dt   - Size of the time step
 */
__global__ void calculateVelocity(Particles pIn, Particles pOut, const unsigned N, float dt)
{
  /********************************************************************************************************************/
  /*  TODO: CUDA kernel to calculate new particles velocity and position, use shared memory to minimize memory access */
  /********************************************************************************************************************/

  unsigned i = blockDim.x * blockIdx.x + threadIdx.x;
  const float posx  = pIn.posx[i];
  const float posy  = pIn.posy[i];
  const float posz  = pIn.posz[i];
  const float w     = pIn.w[i];
  const float velx  = pIn.velx[i];
  const float vely  = pIn.vely[i];
  const float velz  = pIn.velz[i];

  float gVelx     = 0.0f;
  float gVely     = 0.0f;
  float gVelz     = 0.0f;

  float cVelx     = 0.0f;
  float cVely     = 0.0f;
  float cVelz     = 0.0f;

  // filled shared memory will look like this:
  // [ posx1 .. posxB posy1 .. posyB posz1 .. poszB w1 .. wB velx1 .. velxB vely1 .. velyB velz1 .. velzB ]
  // where B = blockSize (which is assumed to be divisible by warpSize)

  // I tried shuffling around with memory accesses and also using 2xfloat4,
  // but this approach was faster and made more sense to me
  extern __shared__ float shared[];

  for (unsigned b = 0u; b < N; b += blockDim.x) {
    shared[threadIdx.x                 ] = pIn.posx[b + threadIdx.x];
    shared[threadIdx.x +     blockDim.x] = pIn.posy[b + threadIdx.x];
    shared[threadIdx.x + 2 * blockDim.x] = pIn.posz[b + threadIdx.x];
    shared[threadIdx.x + 3 * blockDim.x] = pIn.w   [b + threadIdx.x];
    shared[threadIdx.x + 4 * blockDim.x] = pIn.velx[b + threadIdx.x];
    shared[threadIdx.x + 5 * blockDim.x] = pIn.vely[b + threadIdx.x];
    shared[threadIdx.x + 6 * blockDim.x] = pIn.velz[b + threadIdx.x];

    __syncthreads();

    for (unsigned j = 0u; j < blockDim.x; j++) {
      const float otherPosx = shared[j                 ];
      const float otherPosy = shared[j +     blockDim.x];
      const float otherPosz = shared[j + 2 * blockDim.x];
      const float otherW    = shared[j + 3 * blockDim.x];
      const float otherVelx = shared[j + 4 * blockDim.x];
      const float otherVely = shared[j + 5 * blockDim.x];
      const float otherVelz = shared[j + 6 * blockDim.x];

      const float dx = otherPosx - posx;
      const float dy = otherPosy - posy;
      const float dz = otherPosz - posz;

      const float r2 = dx * dx + dy * dy + dz * dz;
      const float r = sqrtf(r2) + FLOAT_MIN;
      const float f = G * w * otherW / r2 + FLOAT_MIN;

      // gravitational velocity computation

      gVelx += (r > COLLISION_DISTANCE) ? dx / r * f : 0.f;
      gVely += (r > COLLISION_DISTANCE) ? dy / r * f : 0.f;
      gVelz += (r > COLLISION_DISTANCE) ? dz / r * f : 0.f;

      // collision velocity computation

      cVelx += (r > 0.f && r < COLLISION_DISTANCE)
                  ? ((((w - otherW) * velx + 2.f * otherW * otherVelx) / (w + otherW)) - velx)
                  : 0.f;
      cVely += (r > 0.f && r < COLLISION_DISTANCE)
                  ? ((((w - otherW) * vely + 2.f * otherW * otherVely) / (w + otherW)) - vely)
                  : 0.f;
      cVelz += (r > 0.f && r < COLLISION_DISTANCE)
                  ? ((((w - otherW) * velz + 2.f * otherW * otherVelz) / (w + otherW)) - velz)
                  : 0.f;
    }

    __syncthreads();
  }
  gVelx *= dt / w;
  gVely *= dt / w;
  gVelz *= dt / w;

  // merging the velocity results and updating particle positions
  // note that we cannot simply perform pOut.pos += ... or pOut.vel += ...
  // since the pIn and pOut data differ after the first iteration

  const float outVelx = velx + gVelx + cVelx;
  const float outVely = vely + gVely + cVely;
  const float outVelz = velz + gVelz + cVelz;

  pOut.posx[i] = posx + outVelx * dt;
  pOut.posy[i] = posy + outVely * dt;
  pOut.posz[i] = posz + outVelz * dt;
  pOut.velx[i] = outVelx;
  pOut.vely[i] = outVely;
  pOut.velz[i] = outVelz;
  
}// end of calculate_gravitation_velocity
//----------------------------------------------------------------------------------------------------------------------

/**
 * CUDA kernel to calculate particles center of mass
 * @param p    - particles
 * @param com  - pointer to a center of mass
 * @param lock - pointer to a user-implemented lock
 * @param N    - Number of particles
 */
__global__ void centerOfMass(Particles p, float4* com, int* lock, const unsigned N)
{

}// end of centerOfMass
//----------------------------------------------------------------------------------------------------------------------

/**
 * CPU implementation of the Center of Mass calculation
 * @param particles - All particles in the system
 * @param N         - Number of particles
 */
__host__ float4 centerOfMassRef(MemDesc& memDesc)
{
  float4 com{};

  for (std::size_t i{}; i < memDesc.getDataSize(); i++)
  {
    const float3 pos = {memDesc.getPosX(i), memDesc.getPosY(i), memDesc.getPosZ(i)};
    const float  w   = memDesc.getWeight(i);

    // Calculate the vector on the line connecting current body and most recent position of center-of-mass
    // Calculate weight ratio only if at least one particle isn't massless
    const float4 d = {pos.x - com.x,
                      pos.y - com.y,
                      pos.z - com.z,
                      ((memDesc.getWeight(i) + com.w) > 0.0f)
                        ? ( memDesc.getWeight(i) / (memDesc.getWeight(i) + com.w))
                        : 0.0f};

    // Update position and weight of the center-of-mass according to the weight ration and vector
    com.x += d.x * d.w;
    com.y += d.y * d.w;
    com.z += d.z * d.w;
    com.w += w;
  }

  return com;
}// enf of centerOfMassRef
//----------------------------------------------------------------------------------------------------------------------
