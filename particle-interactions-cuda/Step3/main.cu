#include "hip/hip_runtime.h"
/**
 * @file      main.cu
 *
 * @author    Ján Maťufka \n
 *            Faculty of Information Technology \n
 *            Brno University of Technology \n
 *            xmatuf00@stud.fit.vutbr.cz
 *
 * @brief     PCG Assignment 1
 *
 * @version   2024
 *
 * @date      04 October   2023, 09:00 (created) \n
 */

#include <cmath>
#include <cstdio>
#include <chrono>
#include <string>

#include "nbody.cuh"
#include "h5Helper.h"

/**
 * @brief CUDA error checking macro
 * @param call CUDA API call
 */
#define CUDA_CALL(call) \
  do { \
    const hipError_t _error = (call); \
    if (_error != hipSuccess) \
    { \
      std::fprintf(stderr, "CUDA error (%s:%d): %s\n", __FILE__, __LINE__, hipGetErrorString(_error)); \
      std::exit(EXIT_FAILURE); \
    } \
  } while(0)

/**
 * Main rotine
 * @param argc
 * @param argv
 * @return
 */
int main(int argc, char **argv)
{
  if (argc != 10)
  {
    std::printf("Usage: nbody <N> <dt> <steps> <threads/block> <write intesity> <reduction threads> <reduction threads/block> <input> <output>\n");
    std::exit(1);
  }

  // Number of particles
  const unsigned N                   = static_cast<unsigned>(std::stoul(argv[1]));
  // Length of time step
  const float    dt                  = std::stof(argv[2]);
  // Number of steps
  const unsigned steps               = static_cast<unsigned>(std::stoul(argv[3]));
  // Number of thread blocks
  const unsigned simBlockDim         = static_cast<unsigned>(std::stoul(argv[4]));
  // Write frequency
  const unsigned writeFreq           = static_cast<unsigned>(std::stoul(argv[5]));
  // number of reduction threads
  const unsigned redTotalThreadCount = static_cast<unsigned>(std::stoul(argv[6]));
  // Number of reduction threads/blocks
  const unsigned redBlockDim         = static_cast<unsigned>(std::stoul(argv[7]));

  // Size of the simulation CUDA grid - number of blocks
  const unsigned simGridDim = (N + simBlockDim - 1) / simBlockDim;
  // Size of the reduction CUDA grid - number of blocks
  const unsigned redGridDim = (redTotalThreadCount + redBlockDim - 1) / redBlockDim;

  // Log benchmark setup
  std::printf("       NBODY GPU simulation\n"
              "N:                       %u\n"
              "dt:                      %f\n"
              "steps:                   %u\n"
              "threads/block:           %u\n"
              "blocks/grid:             %u\n"
              "reduction threads/block: %u\n"
              "reduction blocks/grid:   %u\n",
              N, dt, steps, simBlockDim, simGridDim, redBlockDim, redGridDim);

  const std::size_t recordsCount = (writeFreq > 0) ? (steps + writeFreq - 1) / writeFreq : 0;

  Particles hParticles{};
  float4*   hCenterOfMass{};

  /********************************************************************************************************************/
  /*                              TODO: CPU side memory allocation (pinned)                                           */
  /********************************************************************************************************************/

  hParticles.posx = static_cast<float*>(operator new[](N * sizeof(float)));
  hParticles.posy = static_cast<float*>(operator new[](N * sizeof(float)));
  hParticles.posz = static_cast<float*>(operator new[](N * sizeof(float)));
  hParticles.w    = static_cast<float*>(operator new[](N * sizeof(float)));
  hParticles.velx = static_cast<float*>(operator new[](N * sizeof(float)));
  hParticles.vely = static_cast<float*>(operator new[](N * sizeof(float)));
  hParticles.velz = static_cast<float*>(operator new[](N * sizeof(float)));

  hCenterOfMass = static_cast<float4*>(operator new(sizeof(float4)));

  /********************************************************************************************************************/
  /*                              TODO: Fill memory descriptor layout                                                 */
  /********************************************************************************************************************/
  /*
   * Caution! Create only after CPU side allocation
   * parameters:
   *                            Stride of two            Offset of the first
   *       Data pointer       consecutive elements        element in FLOATS,
   *                          in FLOATS, not bytes            not bytes
  */
  MemDesc md((float *) hParticles.posx,  1,                       0,  // pos_x
             (float *) hParticles.posy,  1,                       0,  // pos_y
             (float *) hParticles.posz,  1,                       0,  // pos_z
             (float *) hParticles.velx,  1,                       0,  // vel_x
             (float *) hParticles.vely,  1,                       0,  // vel_y
             (float *) hParticles.velz,  1,                       0,  // vel_z
             (float *) hParticles.w,     1,                       0,  // weight
             N,
             recordsCount);

  // Initialisation of helper class and loading of input data
  H5Helper h5Helper(argv[8], argv[9], md);

  try
  {
    h5Helper.init();
    h5Helper.readParticleData();
  }
  catch (const std::exception& e)
  {
    std::fprintf(stderr, "Error: %s\n", e.what());
    return EXIT_FAILURE;
  }

  Particles dParticles[2]{};
  float4*   dCenterOfMass{};
  int*      dLock{};

  /********************************************************************************************************************/
  /*                                     TODO: GPU side memory allocation                                             */
  /********************************************************************************************************************/

  for (unsigned i = 0; i < 2; i++) {
    CUDA_CALL(hipMalloc(&(dParticles[i].posx), sizeof(float) * N));
    CUDA_CALL(hipMalloc(&(dParticles[i].posy), sizeof(float) * N));
    CUDA_CALL(hipMalloc(&(dParticles[i].posz), sizeof(float) * N));
    CUDA_CALL(hipMalloc(&(dParticles[i].w),    sizeof(float) * N));
    CUDA_CALL(hipMalloc(&(dParticles[i].velx), sizeof(float) * N));
    CUDA_CALL(hipMalloc(&(dParticles[i].vely), sizeof(float) * N));
    CUDA_CALL(hipMalloc(&(dParticles[i].velz), sizeof(float) * N));
  }

  CUDA_CALL(hipMalloc(&dCenterOfMass, sizeof(float4)));
  CUDA_CALL(hipMalloc(&dLock, sizeof(int)));

  /********************************************************************************************************************/
  /*                                     TODO: Memory transfer CPU -> GPU                                             */
  /********************************************************************************************************************/

  for (unsigned i = 0; i < 2; i++) {
    CUDA_CALL(hipMemcpy(dParticles[i].posx, hParticles.posx, sizeof(float) * N, hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(dParticles[i].posy, hParticles.posy, sizeof(float) * N, hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(dParticles[i].posz, hParticles.posz, sizeof(float) * N, hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(dParticles[i].w,    hParticles.w,    sizeof(float) * N, hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(dParticles[i].velx, hParticles.velx, sizeof(float) * N, hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(dParticles[i].vely, hParticles.vely, sizeof(float) * N, hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(dParticles[i].velz, hParticles.velz, sizeof(float) * N, hipMemcpyHostToDevice));
  }

  /********************************************************************************************************************/
  /*                                     TODO: Clear GPU center of mass                                               */
  /********************************************************************************************************************/

  CUDA_CALL(hipMemset(dCenterOfMass, 0, sizeof(float4)));
  CUDA_CALL(hipMemset(dLock, 0, sizeof(int))); // 0 = unlocked, 1 = locked

  // Get CUDA device warp size
  int device;
  int warpSize;

  CUDA_CALL(hipGetDevice(&device));
  CUDA_CALL(hipDeviceGetAttribute(&warpSize, hipDeviceAttributeWarpSize, device));

  /********************************************************************************************************************/
  /*                                  TODO: Set dynamic shared memory computation                                     */
  /********************************************************************************************************************/
  const std::size_t sharedMemSize    = simBlockDim * sizeof(float) * 7;
  const std::size_t redSharedMemSize = ((redBlockDim + warpSize - 1) / warpSize) * sizeof(float4);   // you can use warpSize variable

  // Start measurement
  const auto start = std::chrono::steady_clock::now();

  for (unsigned s = 0u; s < steps; ++s)
  {
    const unsigned srcIdx = s % 2;        // source particles index
    const unsigned dstIdx = (s + 1) % 2;  // destination particles index

    /******************************************************************************************************************/
    /*                   TODO: GPU kernel invocation with correctly set dynamic memory size                           */
    /******************************************************************************************************************/
    calculateVelocity<<<simGridDim, simBlockDim, sharedMemSize>>>(dParticles[srcIdx], dParticles[dstIdx], N, dt);
  }

  const unsigned resIdx = steps % 2;    // result particles index

  /********************************************************************************************************************/
  /*                                 TODO: Invocation of center of mass kernel                                        */
  /********************************************************************************************************************/
  centerOfMass<<<redGridDim, redBlockDim, redSharedMemSize>>>(dParticles[resIdx], dCenterOfMass, dLock, N);

  // Wait for all CUDA kernels to finish
  CUDA_CALL(hipDeviceSynchronize());

  // End measurement
  const auto end = std::chrono::steady_clock::now();

  // Approximate simulation wall time
  const float elapsedTime = std::chrono::duration<float>(end - start).count();
  std::printf("Time: %f s\n", elapsedTime);

  /********************************************************************************************************************/
  /*                                     TODO: Memory transfer GPU -> CPU                                             */
  /********************************************************************************************************************/

  CUDA_CALL(hipMemcpy(hParticles.posx, dParticles[resIdx].posx, sizeof(float) * N, hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(hParticles.posy, dParticles[resIdx].posy, sizeof(float) * N, hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(hParticles.posz, dParticles[resIdx].posz, sizeof(float) * N, hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(hParticles.w,    dParticles[resIdx].w,    sizeof(float) * N, hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(hParticles.velx, dParticles[resIdx].velx, sizeof(float) * N, hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(hParticles.vely, dParticles[resIdx].vely, sizeof(float) * N, hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(hParticles.velz, dParticles[resIdx].velz, sizeof(float) * N, hipMemcpyDeviceToHost));

  CUDA_CALL(hipMemcpy(hCenterOfMass, dCenterOfMass, sizeof(float4), hipMemcpyDeviceToHost));

  // Compute reference center of mass on CPU
  const float4 refCenterOfMass = centerOfMassRef(md);

  std::printf("Reference center of mass: %f, %f, %f, %f\n",
              refCenterOfMass.x,
              refCenterOfMass.y,
              refCenterOfMass.z,
              refCenterOfMass.w);

  std::printf("Center of mass on GPU: %f, %f, %f, %f\n",
              hCenterOfMass->x,
              hCenterOfMass->y,
              hCenterOfMass->z,
              hCenterOfMass->w);

  // Writing final values to the file
  h5Helper.writeComFinal(*hCenterOfMass);
  h5Helper.writeParticleDataFinal();

  /********************************************************************************************************************/
  /*                                     TODO: GPU side memory deallocation                                           */
  /********************************************************************************************************************/

  for (unsigned i = 0; i < 2; i++) {
    CUDA_CALL(hipFree(dParticles[i].posx));
    CUDA_CALL(hipFree(dParticles[i].posy));
    CUDA_CALL(hipFree(dParticles[i].posz));
    CUDA_CALL(hipFree(dParticles[i].w));
    CUDA_CALL(hipFree(dParticles[i].velx));
    CUDA_CALL(hipFree(dParticles[i].vely));
    CUDA_CALL(hipFree(dParticles[i].velz));
  }

  CUDA_CALL(hipFree(dCenterOfMass));
  CUDA_CALL(hipFree(dLock));

  /********************************************************************************************************************/
  /*                                     TODO: CPU side memory deallocation                                           */
  /********************************************************************************************************************/

  operator delete[](hParticles.posx);
  operator delete[](hParticles.posy);
  operator delete[](hParticles.posz);
  operator delete[](hParticles.w);
  operator delete[](hParticles.velx);
  operator delete[](hParticles.vely);
  operator delete[](hParticles.velz);

  operator delete(hCenterOfMass);

}// end of main
//----------------------------------------------------------------------------------------------------------------------
